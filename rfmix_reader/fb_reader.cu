#include "hip/hip_runtime.h"
/*
 * Adapted from the `_bed_reader.h` script in the `pandas-plink` package.
 * Source: https://github.com/limix/pandas-plink/blob/main/pandas_plink/_bed_reader.h
 */

#include <math.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

#define MIN(a,b) ((a > b) ? b : a)

__global__
void read_fb_chunk_kernel(uint8_t *buff, uint64_t nrows, uint64_t ncols,
		          uint64_t row_start, uint64_t col_start, uint64_t row_end,
			  uint64_t col_end, uint8_t *out, uint64_t *strides,
			  uint64_t row_size) {
    int r = blockIdx.y * blockDim.y + threadIdx.y + row_start;
    int c = blockIdx.x * blockDim.x + threadIdx.x + col_start;

    if (r < row_end && c < col_end) {
        uint64_t buff_index = r * row_size + c / 4;
	char b = buff[buff_index];
	char b0 = b & 0x55;
	char b1 = (b & 0xAA) >> 1;
	char p0 = b0 ^ b1;
	char p1 = (b0 | b1) & b0;
	p1 <<= 1;
	p0 |= p1;
	uint64_t ce = MIN(c + 4, col_end);

        for (; c < ce; ++c) {
	    out[(r - row_start) * strides[0] + (c - col_start) * strides[1]] = p0 & 3;
	    p0 >>= 2;
	}
    }
}

void read_fb_chunk(uint8_t *buff, uint64_t nrows, uint64_t ncols,
		   uint64_t row_start, uint64_t col_start, uint64_t row_end,
		   uint64_t col_end, uint8_t *out, uint64_t *strides) {
    uint64_t row_size = (ncols + 3) / 4;		   

    // Allocate GPU memory
    uint8_t *d_buff, *d_out;
    uint64_t *d_strides;
    hipMalloc(&d_buff, nrows * row_size * sizeof(uint8_t));
    hipMalloc(&d_out, (row_end - row_start) * (col_end - col_start) * sizeof(uint8_t));
    hipMalloc(&d_strides, 2 * sizeof(uint64_t));

    // Copy data to GPU
    hipMemcpy(d_buff, buff, nrows * row_size * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_strides, strides, 2 * sizeof(uint64_t), hipMemcpyHostToDevice)

    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((col_end - col_start + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (row_end - row_start + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    read_fb_chunk_kernel<<<numBlocks, threadsPerBlock>>>(d_buff, nrows, ncols, row_start, col_start, row_end, col_end, d_out, d_strides, row_size);

    // Copy results back to host
    hipMemcpy(out, d_out, (row_end - row_start) * (col_end - col_start) * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_buff);
    hipFree(d_out);
    hipFree(d_strides);
}
